#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <boost/timer/timer.hpp>

__global__ void kernel(int * a, int * b, int count) {
  int offset = threadIdx.x + blockDim.x * blockIdx.x;
  for (int index = offset; index < count; index += blockDim.x * gridDim.x) {
    b[index] = 2 * a[index];
  }
}

std::vector<int> ver1(const std::vector<int> &vec, const int N) {
  int *a_d = nullptr;
  hipMalloc((void**)&a_d, sizeof(int) * N);
  int *b_d = nullptr;
  hipMalloc((void**)&b_d, sizeof(int) * N);
  hipMemcpy(a_d, vec.data(), sizeof(int) * N, hipMemcpyHostToDevice);
  kernel<<<1024, 256>>>(a_d, b_d, N);
  std::vector<int> b(N);
  hipMemcpy(b.data(), b_d, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(b_d);
  return b;
}

std::vector<int> ver2(const std::vector<int> &vec, const int N, const int n) {
  int *a_h = nullptr;
  hipHostMalloc((void**)&a_h, sizeof(int) * N);
  int *b_h = nullptr;
  hipHostMalloc((void**)&b_h, sizeof(int) * N);
  int *a_d = nullptr;
  hipMalloc((void**)&a_d, sizeof(int) * N);
  int *b_d = nullptr;
  hipMalloc((void**)&b_d, sizeof(int) * N);
  memcpy(a_h, vec.data(), sizeof(int) * N);
  hipStream_t str[3];
  for (int i = 0; i < 3; ++i) {
    hipStreamCreate(str + i);
  }
  for (int i = 0; i < 3; ++i) {
    hipMemcpyAsync(a_d + n*i, a_h + n*i, sizeof(int) * n, hipMemcpyHostToDevice, str[i]);
  }
  for (int i = 0; i < 3; ++i) {
    kernel<<<1024, 256, 0, str[i]>>>(a_d + n*i, b_d + n*i, n);
  }
  for (int i = 0; i < 3; ++i) {
    hipMemcpyAsync(b_h + n*i, b_d + n*i, sizeof(int) * n, hipMemcpyDeviceToHost, str[i]);
  }
  for (int i = 0; i < 3; ++i) {
    hipStreamSynchronize(str[i]);
    hipStreamDestroy(str[i]);
  }
  std::vector<int> b(N);
  memcpy(b.data(), b_h, sizeof(int) * N);
  hipFree(a_h);
  hipFree(b_h);
  hipFree(a_d);
  hipFree(b_d);
  return b;
}

int main(int argc, char **argv) {
  if (argc < 3) {
    std::cerr << "Usage: " << argv[0] << " N K" << std::endl;
    return -1;
  }
  int n = atoi(argv[1]);
  int k = atoi(argv[2]);
  int N = 3 * n;
  std::vector<int> vec(N);
  std::random_device rd;
  std::mt19937 mt(rd());
  for (int i = 0; i < N; ++i) {
    vec[i] = mt();
  }
  boost::timer::cpu_timer timer;
  std::vector<int> b;
  switch (k) {
    case 0: b = ver1(vec, N); break;
    case 1: b = ver2(vec, N, n); break;
  }
  std::cout << timer.format() << std::endl;
  int diff = 0;
  for (int i = 0; i < N; ++i) {
    diff += abs(vec[i] * 2 - b[i]);
  }
  std::cerr << "diff: " << diff << std::endl;
  return 0;
}
